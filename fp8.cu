#include "gemm.h"

using namespace cute;

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::mx_float8_t<cutlass::float_e4m3_t>;    // Element type for A matrix operand
using         LayoutATag  = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 16;                                             // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = cutlass::mx_float4_t<cutlass::float_e2m1_t>;    // Element type for B matrix operand
using         LayoutBTag  = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 32;                                             // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementD    = cutlass::bfloat16_t;                            // Element type for D matrix operand
using         ElementC    = cutlass::bfloat16_t;                            // Element type for C matrix operand
using         LayoutCTag  = cutlass::layout::RowMajor;                      // Layout type for C matrix operand
using         LayoutDTag  = cutlass::layout::RowMajor;                      // Layout type for D matrix operand
constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)
// Kernel functional config
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm120;                           // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassBlockScaledTensorOp;      // Operator class tag

// Kernel Perf config
using ThreadBlockShape    = Shape<_128,_128,_128>;                          // Threadblock's tile size
using ClusterShape        = Shape<_1,_1,_1>;                                // Shape of the threadblocks in a cluster

float matmul_host8(
        const ElementA::DataType *A,
        const ElementB::DataType *B,
        int M,
        int N,
        int K,
        ElementC *C,
        ElementD *D,
        const ElementA::ScaleFactorType *SFA,
        const ElementB::ScaleFactorType *SFB
)
{

    using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
        ArchTag, OperatorClass,                      
        ThreadBlockShape, ClusterShape,
        cutlass::epilogue::collective::EpilogueTileAuto,
        ElementAccumulator, ElementAccumulator,
        ElementC, LayoutCTag, AlignmentC,
        ElementD, LayoutDTag, AlignmentD,
        cutlass::epilogue::collective::EpilogueScheduleAuto                      // Epilogue schedule policy
        >::CollectiveOp;

    using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
        ArchTag, OperatorClass,
        ElementA, LayoutATag, AlignmentA,
        ElementB, LayoutBTag, AlignmentB,
        ElementAccumulator,
        ThreadBlockShape, ClusterShape,
        cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
        cutlass::gemm::collective::KernelScheduleAuto                             // Kernel schedule policy. Auto defaults to cooperative kernel schedule
        >::CollectiveOp;

    using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
        Shape<int,int,int,int>,                                                   // Indicates ProblemShape
        CollectiveMainloop,
        CollectiveEpilogue,
        void>;

    using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

    // Reference device GEMM implementation type
    using StrideA   = typename Gemm::GemmKernel::StrideA;
    using LayoutA   = decltype(cute::make_layout(make_shape(0,0,0), StrideA{}));
    using LayoutSFA = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFA;      // Scale Factor tensors have an interleaved layout. Bring Layout instead of stride.
    using StrideB   = typename Gemm::GemmKernel::StrideB;
    using LayoutB   = decltype(cute::make_layout(make_shape(0,0,0), StrideB{}));
    using LayoutSFB = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFB;      // Scale Factor tensors have an interleaved layout. Bring Layout instead of stride.
    using StrideC   = typename Gemm::GemmKernel::StrideC;
    using LayoutC   = decltype(cute::make_layout(make_shape(0,0,0), StrideC{}));
    using StrideD   = typename Gemm::GemmKernel::StrideD;
    using LayoutD   = decltype(cute::make_layout(make_shape(0,0,0), StrideD{}));

    //
    // Data members
    //

    /// Initialization
    StrideA stride_A;
    LayoutA layout_A;
    LayoutSFA layout_SFA;
    StrideB stride_B;
    LayoutB layout_B;
    LayoutSFB layout_SFB;
    StrideC stride_C;
    LayoutC layout_C;
    StrideD stride_D;
    LayoutD layout_D;

    cutlass::HostTensor<ElementA::DataType, cutlass::layout::PackedVectorLayout> block_A;
    cutlass::HostTensor<ElementA::ScaleFactorType, cutlass::layout::PackedVectorLayout> block_SFA;
    cutlass::HostTensor<ElementB::DataType, cutlass::layout::PackedVectorLayout> block_B;
    cutlass::HostTensor<ElementB::ScaleFactorType, cutlass::layout::PackedVectorLayout> block_SFB;
    cutlass::HostTensor<ElementC, cutlass::layout::PackedVectorLayout> block_C;
    // Output Tensor
    cutlass::HostTensor<ElementD, cutlass::layout::PackedVectorLayout> block_D;

    // For SFA and SFB tensors layouts
    using Sm1xxBlkScaledConfig =  typename Gemm::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;

    stride_A = cutlass::make_cute_packed_stride(StrideA{}, {M, K, 1});
    stride_B = cutlass::make_cute_packed_stride(StrideB{}, {N, K, 1});
    stride_C = cutlass::make_cute_packed_stride(StrideC{}, {M, N, 1});
    stride_D = cutlass::make_cute_packed_stride(StrideD{}, {M, N, 1});

    layout_A = make_layout(make_shape(M, K, 1), stride_A);
    layout_B = make_layout(make_shape(N, K, 1), stride_B);
    layout_C = make_layout(make_shape(M, N, 1), stride_C);
    layout_D = make_layout(make_shape(M, N, 1), stride_D);
    layout_SFA = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFA(cute::make_shape(M, N, K, 1));
    layout_SFB = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFB(cute::make_shape(M, N, K, 1));


    block_A.reset(cutlass::make_Coord(size(layout_A)));
    block_B.reset(cutlass::make_Coord(size(layout_B)));
    block_C.reset(cutlass::make_Coord(size(layout_C)));
    block_D.reset(cutlass::make_Coord(size(layout_D)));
    block_SFA.reset(cutlass::make_Coord(size(filter_zeros(layout_SFA))));
    block_SFB.reset(cutlass::make_Coord(size(filter_zeros(layout_SFB))));

    block_A.copy_in_host_to_device(A);
    block_B.copy_in_host_to_device(B);
    block_C.copy_in_host_to_device(C);
    block_SFA.copy_in_host_to_device(SFA);
    block_SFB.copy_in_host_to_device(SFB);

    block_A.sync_device();
    block_B.sync_device();
    block_C.sync_device();
    block_SFA.sync_device();
    block_SFB.sync_device();
    
    // Timing using CUDA events
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start));
    Gemm gemmOp;

    typename Gemm::Arguments arguments {
        cutlass::gemm::GemmUniversalMode::kGemm,
        {M, N, K, 1},
        { // Mainloop arguments
            block_A.device_data(), stride_A,
            block_B.device_data(), stride_B,
            block_SFA.device_data(), layout_SFA,
            block_SFB.device_data(), layout_SFB
        },
        { // Epilogue arguments
            {1.0, 0},
            block_C.device_data(), stride_C,
            block_D.device_data(), stride_D
        }
    };

    auto status = gemmOp(arguments);

    assert(status == cutlass::Status::kSuccess);
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    return milliseconds;
    // std::printf("GEMM completed in %.3f ms\n", milliseconds);

}

